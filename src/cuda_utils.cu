#include <stdint.h>
#include <stdio.h>
#include "hip/hip_runtime.h"


#define GPU_TOUCH_STEP 4096

__global__ void cuda_touch_pages(volatile uint8_t *c, int size,
		volatile int *stop_flag, int is_infinite) {
	do {
		for (int iter = 0; iter < size; iter += GPU_TOUCH_STEP)
			c[iter] = 0;
	}
	while (is_infinite && !*stop_flag);
}

extern "C" int touch_gpu_pages(uint8_t *addr, int buf_size,
		int is_infinite, volatile int **stop_flag)
{
	hipError_t ret = hipMallocManaged((void **)stop_flag, sizeof(int));
	if (ret) {
		printf("failed to allocate stop flag\n");
		return -1;
	}

	*stop_flag = 0;
	cuda_touch_pages<<<1, 1>>>(addr, buf_size, *stop_flag, is_infinite);

	return 0;
}
